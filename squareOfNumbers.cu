//============================================================================
// Name        : cudaProg.cpp
// Author      : Pratil
// Version     :
// Copyright   : Your copyright notice
// Description : Hello World in C++, Ansi-style
//============================================================================


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
using namespace std;

__global__ void squareFunc(unsigned int *d_in, unsigned int *d_out)
{
    int idx = threadIdx.x;
    unsigned int val = d_in[idx];
    d_out[idx] = val * val;
    //printf("%d square value %d \n  ", idx, d_out[idx]);  
}


int main()
{

    const unsigned int arr_len = 64;
    const unsigned int arr_size = 64 * sizeof(unsigned int);

    unsigned int arr_in[arr_len];
    unsigned int arr_out[arr_len];
    for (unsigned int i = 0; i < 64; i++)
    {
        arr_in[i] = i;
        cout << i << "   :    " << arr_in[i] << endl;
    }

    unsigned int *d_in;
    unsigned int *d_out;

    hipMalloc((void**) &d_in, arr_size);
    hipMalloc((void**) &d_out, arr_size);

    hipMemcpy(d_in, arr_in, arr_size, hipMemcpyHostToDevice);

    squareFunc<<<1,64>>>(d_in, d_out);

    hipMemcpy(arr_out, d_out, arr_size, hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < 64; i++)
    {
       cout << i <<"    :   " << arr_out[i] << endl;
    }
   hipDeviceSynchronize();

    hipFree(d_out);
    hipFree(d_in);
return 0;
}
